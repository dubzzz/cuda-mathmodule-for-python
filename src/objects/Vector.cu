#include "../checks/CudaChecks.hpp"
#include "Vector.hpp"
#include <iostream>
#include "../preproc.hpp"

void init_vector() {
	__LOG__
	import_array();
}

Vector::Vector(const unsigned int &size) : size_(size) {
	__LOG__
	smart_ptr_counter_ = new int(1);
	cudaErrorCheck(hipMalloc(&data_, size_ * sizeof(double)));
}

Vector::Vector(const Vector &v) : size_(v.size_), data_(v.data_), smart_ptr_counter_(v.smart_ptr_counter_) {
	__LOG__
	(*smart_ptr_counter_) += 1;
}

Vector::Vector(const double *h_v, const unsigned int &size) : size_(size) {
	__LOG__
	smart_ptr_counter_ = new int(1);
	cudaErrorCheck(hipMalloc(&data_, size_ * sizeof(double)));
	cudaErrorCheck(hipMemcpy(data_, h_v, size_ * sizeof(double), hipMemcpyHostToDevice));
}

Vector::~Vector() {
	__LOG__
	if (! data_)
		return;
	
	if(*smart_ptr_counter_ > 1) {// cuda-kernel constructs a copy of the object and then call its destructor
		(*smart_ptr_counter_) -= 1;
		return;
	}
	
	delete smart_ptr_counter_;
	cudaErrorCheck(hipFree(data_));
}

void Vector::free() {
	__LOG__
	if(*smart_ptr_counter_ > 1) {// cuda-kernel constructs a copy of the object and then call its destructor
		(*smart_ptr_counter_) -= 1;
		data_ = 0;
		return;
	}
	
	delete smart_ptr_counter_;
	
	cudaErrorCheck(hipFree(data_));
	data_ = 0;
}		

void Vector::memsetZero() {
	__LOG__
	cudaErrorCheck(hipMemset(data_, 0, size_ * sizeof(double)));
}

PyArrayObject *Vector::toNumPy() {
	__LOG__
	int dims[] = {size_};
	PyArrayObject *h_arrayNumPy = (PyArrayObject *) PyArray_FromDims(1, dims, NPY_DOUBLE);
	cudaErrorCheck(hipMemcpy(h_arrayNumPy->data, data_, size_ * sizeof(double), hipMemcpyDeviceToHost));
	return h_arrayNumPy;
}

__device__ double& Vector::get(const unsigned int &x) const {
	return data_[x];
}

__device__ double& Vector::operator[](const unsigned int &x) const {
	return data_[x];
}

__device__ unsigned int Vector::getSize() const { return size_; }

