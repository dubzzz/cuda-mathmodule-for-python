#include <iostream>
#include <stdio.h>

#include "CudaChecks.hpp"

void cudaAssert(const hipError_t err, const char *file, const int line)
{ 
	if(hipSuccess != err)
	{                                                
		fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", file, line, hipGetErrorString(err));
		exit(1);
	} 
}

